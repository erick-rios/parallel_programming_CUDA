/**
 * @file cubicSum.cu
 * @brief Sum of the squares of the first N natural numbers using parallel reduction
 * @details This program calculates the sum of the squares of the first N natural numbers using a parallel reduction
 * algorithm.
 * 
 * The program initializes an array of size N with the squares of the first N natural numbers, then it calculates the sum
 * of the squares of the array using a parallel reduction algorithm. The algorithm uses a two-step reduction in shared
 * memory to calculate the sum of the squares of the array. The program uses a block size of 512 threads and a grid size
 * of (N + 1023) / 1024 blocks.
 * 
 * The program validates the result by comparing it with the expected result of the sum of the squares of the first N
 * natural numbers.
 * 
 * @author Erick Jesús Ríos González
 * @date 05/11/2024
 */
#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>

// Define the size of the array
#define N 1024

/**
 * @brief Function to calculate the sum of the squares of an array using a parallel reduction algorithm
 * @details This function calculates the sum of the squares of an array using a parallel reduction algorithm.
 * 
 * The function uses a two-step reduction in shared memory to calculate the sum of the squares of the array. The function
 * uses a block size of 512 threads and a grid size of (N + 1023) / 1024 blocks.
 * 
 * @param input Array of integers to calculate the sum of the squares
 * @param result Pointer to store the result of the sum of the squares
 * @param n Size of the array
 */
__global__ void squareSumReduction(int *input, int *result, int n) {
    extern __shared__ int sdata[];
    
    // Global thread index
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x * 2 + threadIdx.x;
    
    // Load data into shared memory and perform the first step of the reduction
    sdata[tid] = (i < n) ? input[i] * input[i] : 0;
    if (i + blockDim.x < n) sdata[tid] += input[i + blockDim.x] * input[i + blockDim.x];
    __syncthreads();

    // Reduce the data in shared memory
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // Write the result of the block to global memory
    if (tid == 0) result[blockIdx.x] = sdata[0];
}

/**
 * @brief CPU function to calculate the sum of the squares of an array
 * @details This function calculates the sum of the squares of an array using the CPU.
 * 
 * The function calculates the sum of the squares of the array using a for loop to iterate over the elements of the array
 * and add the square of each element to the sum.
 * 
 * @param data Array of integers to calculate the sum of the squares
 * @param n Size of the array
 */
long long cpuSquareSum(int *data, int n) {
    long long sum = 0;
    for (int i = 0; i < n; i++) {
        sum += data[i] * data[i];
    }
    return sum;
}

/**
 * @brief Main function
 * @details This function initializes the host data, allocates memory on the device, transfers data from the host to the
 * device, and measures the time taken by the GPU and the CPU to calculate the sum of the squares of the numbers from 1
 * to N.
 * 
 * @return 0 on success
 */
int main() {
    // Initialize host data
    int *host_data = new int[N];
    for (int i = 0; i < N; i++) {
        host_data[i] = i + 1;
    }

    // Result variables
    int host_result = 0;
    long long cpu_result = 0;

    // Allocate memory on the device
    int *device_data, *device_partial_sums;
    hipMalloc((void**)&device_data, N * sizeof(int));
    hipMalloc((void**)&device_partial_sums, (N / 1024) * sizeof(int));

    // Transfer data from the host to the device
    hipMemcpy(device_data, host_data, N * sizeof(int), hipMemcpyHostToDevice);

    // Measure time on the GPU
    auto gpu_start = std::chrono::high_resolution_clock::now();
    
    // Set up the kernel configuration
    int threads = 512;
    int blocks = (N + threads * 2 - 1) / (threads * 2);
    squareSumReduction<<<blocks, threads, threads * sizeof(int)>>>(device_data, device_partial_sums, N);

    // Reduce the partial sums
    int *host_partial_sums = new int[blocks];
    hipMemcpy(host_partial_sums, device_partial_sums, blocks * sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i < blocks; i++) {
        host_result += host_partial_sums[i];
    }

    auto gpu_end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> gpu_duration = gpu_end - gpu_start;

    // Measure time on the CPU
    auto cpu_start = std::chrono::high_resolution_clock::now();
    cpu_result = cpuSquareSum(host_data, N);
    auto cpu_end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> cpu_duration = cpu_end - cpu_start;

    // Validate the result
    long long expected_result = (static_cast<long long>(N) * (N + 1) * (2 * N + 1)) / 6;
    if (host_result == expected_result && cpu_result == expected_result) {
        std::cout << "Resultado correcto: " << host_result << std::endl;
    } else {
        std::cout << "Resultado incorrecto: GPU: " << host_result 
                  << ", CPU: " << cpu_result 
                  << ", esperado: " << expected_result << std::endl;
    }

    // Print the time taken by the GPU and the CPU
    std::cout << "Tiempo en GPU: " << gpu_duration.count() << " ms" << std::endl;
    std::cout << "Tiempo en CPU: " << cpu_duration.count() << " ms" << std::endl;
    std::cout << "Speed-up: " << cpu_duration.count() / gpu_duration.count() << std::endl;

    // Free memory
    delete[] host_data;
    delete[] host_partial_sums;
    hipFree(device_data);
    hipFree(device_partial_sums);

    return 0;
}
