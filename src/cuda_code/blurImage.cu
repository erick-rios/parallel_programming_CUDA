#include "hip/hip_runtime.h"
/**
 * @file blurImage.cu
 * @brief This file contains the implementation of a CUDA kernel that applies a blur effect to an image.
 * 
 * The image is loaded from the file system and then copied to the device memory. The kernel is executed
 * to apply the blur effect to the image and the result is copied back to the host memory to be displayed.
 * 
 * The blur effect is applied to each pixel of the image by averaging the pixel values of the surrounding
 * pixels. The size of the blur effect is defined by the BLUR_SIZE constant.
 * 
 * The image is loaded from the file system using the OpenCV library.
 * 
 * @author Erick Jesús Ríos González
 * @date 17/09/2024
 */
#include <iostream>
#include <opencv2/opencv.hpp>
#include <hip/hip_runtime.h>

using namespace std;
using namespace cv;

#define BLUR_SIZE 10  // Size of the blur effect

/**
 * @brief CUDA kernel that applies a blur effect to an image.
 * 
 * @param input Input image data.
 * @param output Output image data.
 * @param width Image width.
 * @param height Image height.
 */
__global__ void blurKernel(unsigned char* input, unsigned char* output, int width, int height) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (col < width && row < height) {
        for (int channel = 0; channel < 3; channel++) {
            int pixelValue = 0;
            int pixels = 0;

            for (int blurRow = -BLUR_SIZE; blurRow < BLUR_SIZE + 1; blurRow++) {
                for (int blurCol = -BLUR_SIZE; blurCol < BLUR_SIZE + 1; blurCol++) {
                    int currentRow = row + blurRow;
                    int currentCol = col + blurCol;

                    if (currentRow > -1 && currentRow < height && currentCol > -1 && currentCol < width) {
                        int offset = (currentRow * width + currentCol) * 3 + channel;
                        pixelValue += input[offset];
                        pixels++;
                    }
                }
            }

            int outputOffset = (row * width + col) * 3 + channel;
            output[outputOffset] = (unsigned char)(pixelValue / pixels);
        }
    }
}

/**
 * @brief Main function.
 * 
 * @return Program exit status.
 */
int main() {
    Mat image = imread("../../images/5c05de636f596cb157698cde7923ce19e8473211228abb1cea24a12baaaa8074.jpg", IMREAD_COLOR);
    if (image.empty()) {
        cerr << "Error: No se pudo cargar la imagen" << endl;
        return exit(-1);
    }

    int width = image.cols;
    int height = image.rows;

    unsigned char* hostInputImage = image.data;
    unsigned char* hostOutputImage = (unsigned char*)malloc(width * height * 3 * sizeof(unsigned char));

    unsigned char *deviceInputImage, *deviceOutputImage;
    hipMalloc((void**)&deviceInputImage, width * height * 3 * sizeof(unsigned char));
    hipMalloc((void**)&deviceOutputImage, width * height * 3 * sizeof(unsigned char));

    hipMemcpy(deviceInputImage, hostInputImage, width * height * 3 * sizeof(unsigned char), hipMemcpyHostToDevice);

    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    blurKernel<<<gridSize, blockSize>>>(deviceInputImage, deviceOutputImage, width, height);
    hipDeviceSynchronize();

    hipMemcpy(hostOutputImage, deviceOutputImage, width * height * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost);

    Mat outputImage(height, width, CV_8UC3, hostOutputImage);

    imshow("Original Image", image);
    imshow("Blurred Image", outputImage);
    waitKey(0);

    free(hostOutputImage);
    hipFree(deviceInputImage);
    hipFree(deviceOutputImage);

    return 0;
}
