/**
 * @file vectorialSum.cu
 * @brief CUDA program to perfom vectorial subtraction on the GPU.
 * 
 * This program performs vectorial subtraction on the GPU. The program takes two vectors as input and returns the sum of the two vectors.
 * Initialize the vectors with random values and print the result.
 * 
 * @author ERICK JESUS RIOS GONZALEZ
 * @date 25/08/2024
 * @version 1.0
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>

#define BLOCK_SIZE 2048

/**
 * @brief Function to initialize the vector with random values.
 * 
 * This function initializes the vector with random values between 1 and 100
 * 
 * @param vectorToInit Pointer to the vector to be initialized.
 * @param size Size of the vector.  
 */

void initVector(float *vectorToInit, int size) {
    for (int i = 0; i < size; i++) {
        vectorToInit[i] = (float)rand() / RAND_MAX;
    }
}

/**
 * @brief Function to show the vector.
 * 
 * This function prints the vector.
 * 
 * @param vectorToPrint Pointer to the vector to be printed.
 * @param size Size of the vector.  
 */
void showVector(float *vectorToPrint, int size) {
    for (int i = 0; i < size; i++) {
        printf("V[%d] = %f\n", i, vectorToPrint[i]);
    }
    printf("\n");
}

/**
 * @brief Function to perform vectorial subtraction on the GPU.
 * This function performs vectorial subtraction on the GPU.
 * @param vectorA Pointer to the first vector.
 * @param vectorB Pointer to the second vector.
 * @param vectorC Pointer to the resulting vector.
 * @param size Size of the vectors. 
 */
__global__ void vectorialSubtraction(float *vectorA, float *vectorB, float *vectorC, int size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size) {
        vectorC[index] = vectorA[index] - vectorB[index];
    }
}

/**
 * @brief Main function.
 * 
 * This function initializes the vectors, allocates memory on the GPU, copies the vectors to the GPU, performs the vectorial subtraction, copies the result to the CPU, and prints the result.
 * @return 0 if the program ends correctly.
 */

int main(){
    int size = 1 << 20;

    // Allocate memory on the CPU, define pointers to the vectors, and initialize the vectors.
    float *vectorA, *vectorB, *vectorC;
    // Allocate memory on the GPU, define pointers to the vectors.
    float *d_vectorA, *d_vectorB, *d_vectorC;
    struct timeval start, end;

    // Allocate memory on the CPU.
    vectorA = (float *)malloc(size * sizeof(float));
    vectorB = (float *)malloc(size * sizeof(float));
    vectorC = (float *)malloc(size * sizeof(float));

    // Initialize the vectors.
    initVector(vectorA, size);
    initVector(vectorB, size);

    // Allocate memory on the GPU.
    hipMalloc(&d_vectorA, size * sizeof(float));
    hipMalloc(&d_vectorB, size * sizeof(float));
    hipMalloc(&d_vectorC, size * sizeof(float));

    // Initialize the timer.
    gettimeofday(&start, NULL);

    // Copy the vectors to the GPU.
    hipMemcpy(d_vectorA, vectorA, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_vectorB, vectorB, size * sizeof(float), hipMemcpyHostToDevice);

    // Perform the vectorial subtraction.
    vectorialSubtraction<<<(size + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(d_vectorA, d_vectorB, d_vectorC, size);

    // Copy the result to the CPU.
    hipMemcpy(vectorC, d_vectorC, size * sizeof(float), hipMemcpyDeviceToHost);

    // Stop the timer.
    gettimeofday(&end, NULL);
    
    // Print the time.
    printf("Time: %ld microseconds\n", (end.tv_sec - start.tv_sec) * 1000000 + end.tv_usec - start.tv_usec);

    // Print the result.
    //showVector(vectorC, size);
    //showVector(vectorA, size);
    //showVector(vectorB, size);

    // Free memory on the CPU
    free(vectorA);
    free(vectorB);
    free(vectorC);

    // Free memory on the GPU
    hipFree(d_vectorA);
    hipFree(d_vectorB);
    hipFree(d_vectorC);

    return 0;
}